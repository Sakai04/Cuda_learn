
#include <hip/hip_runtime.h>
/*#include "cuda_runtime.h"
#include "device_launch_parameters.h"
#include <stdio.h>

__global__ void print_data(int* _dDataPtr)
{
    printf("%d ", _dDataPtr[threadIdx.x]);
}

__global__ void set_data(int* _dDataPtr)
{
    _dDataPtr[threadIdx.x] = 2;
}

int main(void)
{
    int data[10] = { 0 };
    for (int i = 0; i < 10; i++) data[i] = 1;

    int* dDataPtr;
    cudaError_t errorCode;

    errorCode = cudaMalloc(&dDataPtr, sizeof(int) * 10);
    if (errorCode != cudaSuccess)
    {
        fprintf(stderr, "cudaMalloc failed: %s\n", cudaGetErrorString(errorCode));
        return -1;
    }

    errorCode = cudaMemcpy(dDataPtr, data, sizeof(int) * 10, cudaMemcpyHostToDevice);
    if (errorCode != cudaSuccess)
    {
        fprintf(stderr, "cudaMemcpy (Host to Device) failed: %s\n", cudaGetErrorString(errorCode));
        cudaFree(dDataPtr);
        return -1;
    }

    set_data<<<1, 10>>>(dDataPtr);
    cudaDeviceSynchronize();

    errorCode = cudaMemcpy(data, dDataPtr, sizeof(int) * 10, cudaMemcpyDeviceToHost);
    if (errorCode != cudaSuccess)
    {
        fprintf(stderr, "cudaMemcpy (Device to Host) failed: %s\n", cudaGetErrorString(errorCode));
        cudaFree(dDataPtr);
        return -1;
    }

    print_data<<<1, 10>>>(dDataPtr);
    cudaDeviceSynchronize();

    errorCode = cudaFree(dDataPtr);
    if (errorCode != cudaSuccess)
    {
        fprintf(stderr, "cudaFree failed: %s\n", cudaGetErrorString(errorCode));
        return -1;
    }

    return 0;
}*/