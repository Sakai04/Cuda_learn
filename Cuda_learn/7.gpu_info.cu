#include "hip/hip_runtime.h"


#include <stdio.h>

#define _1MB (1024 * 1024)

// SM ������ �ھ� ���� ��ȯ�ϴ� �Լ�
int ConvertSMVer2Cores(int major, int minor)
{
    // GPU ��Ű��ó ������ ���� (SM ������ ����Ͽ� SM�� �ھ� ���� ����)
    typedef struct {
        int SM; // 0xMm (16���� ǥ���), M = SM �ֿ� ����, m = SM �� ����
        int Cores;
    } sSMtoCores;

    sSMtoCores nGpuArchCoresPerSM[] = {
        { 0x30, 192 }, // Kepler ���� (SM 3.0) GK10x Ŭ����
        { 0x32, 192 }, // Kepler ���� (SM 3.2) GK10x Ŭ����
        { 0x35, 192 }, // Kepler ���� (SM 3.5) GK11x Ŭ����
        { 0x37, 192 }, // Kepler ���� (SM 3.7) GK21x Ŭ����
        { 0x50, 128 }, // Maxwell ���� (SM 5.0) GM10x Ŭ����
        { 0x52, 128 }, // Maxwell ���� (SM 5.2) GM20x Ŭ����
        { 0x53, 128 }, // Maxwell ���� (SM 5.3) GM20x Ŭ����
        { 0x60, 64  }, // Pascal ���� (SM 6.0) GP100 Ŭ����
        { 0x61, 128 }, // Pascal ���� (SM 6.1) GP10x Ŭ����
        { 0x62, 128 }, // Pascal ���� (SM 6.2) GP10x Ŭ����
        { 0x70, 64  }, // Volta ���� (SM 7.0) GV100 Ŭ����
        { 0x72, 64  }, // Volta ���� (SM 7.2) GV11b Ŭ����
        { 0x75, 64  }, // Turing ���� (SM 7.5) TU10x Ŭ����
        { 0x80, 64  }, // Ampere ���� (SM 8.0) GA100 Ŭ����
        { 0x86, 128 }, // Ampere ���� (SM 8.6) GA10x Ŭ����
        { -1, -1 }
    };

    int index = 0;

    while (nGpuArchCoresPerSM[index].SM != -1) {
        if (nGpuArchCoresPerSM[index].SM == ((major << 4) + minor)) {
            return nGpuArchCoresPerSM[index].Cores;
        }
        index++;
    }

    // ���� ã�� ���� ���, ���� ���� ����Ͽ� ����� ����ǵ��� �⺻���� ���
    printf("SM %d.%d�� ���� MapSMtoCores�� ���ǵ��� �ʾҽ��ϴ�. �⺻������ %d Cores/SM�� ����մϴ�\n", major, minor, nGpuArchCoresPerSM[13].Cores);
    return nGpuArchCoresPerSM[13].Cores;
}

int main(void)
{
    int ngpus;
    hipGetDeviceCount(&ngpus);

    for (int i = 0; i < ngpus; i++)
    {
        hipDeviceProp_t devProp;
        hipGetDeviceProperties(&devProp, i);

        printf("Device %d: %s\n", i, devProp.name);
        printf("\tCompute Capability: %d.%d\n", devProp.major, devProp.minor);
        printf("\tCUDA �ھ� ��: %d\n", ConvertSMVer2Cores(devProp.major, devProp.minor) * devProp.multiProcessorCount);
        printf("\t�۷ι� �޸� ũ��: %.2f MB\n", (float)devProp.totalGlobalMem / _1MB);
    }

    return 0;
}