#include "hip/hip_runtime.h"


#include <stdio.h>

#define _1MB (1024 * 1024)

// SM 버전을 코어 수로 변환하는 함수
int ConvertSMVer2Cores(int major, int minor)
{
    // GPU 아키텍처 유형을 정의 (SM 버전을 사용하여 SM당 코어 수를 결정)
    typedef struct {
        int SM; // 0xMm (16진수 표기법), M = SM 주요 버전, m = SM 부 버전
        int Cores;
    } sSMtoCores;

    sSMtoCores nGpuArchCoresPerSM[] = {
        { 0x30, 192 }, // Kepler 세대 (SM 3.0) GK10x 클래스
        { 0x32, 192 }, // Kepler 세대 (SM 3.2) GK10x 클래스
        { 0x35, 192 }, // Kepler 세대 (SM 3.5) GK11x 클래스
        { 0x37, 192 }, // Kepler 세대 (SM 3.7) GK21x 클래스
        { 0x50, 128 }, // Maxwell 세대 (SM 5.0) GM10x 클래스
        { 0x52, 128 }, // Maxwell 세대 (SM 5.2) GM20x 클래스
        { 0x53, 128 }, // Maxwell 세대 (SM 5.3) GM20x 클래스
        { 0x60, 64  }, // Pascal 세대 (SM 6.0) GP100 클래스
        { 0x61, 128 }, // Pascal 세대 (SM 6.1) GP10x 클래스
        { 0x62, 128 }, // Pascal 세대 (SM 6.2) GP10x 클래스
        { 0x70, 64  }, // Volta 세대 (SM 7.0) GV100 클래스
        { 0x72, 64  }, // Volta 세대 (SM 7.2) GV11b 클래스
        { 0x75, 64  }, // Turing 세대 (SM 7.5) TU10x 클래스
        { 0x80, 64  }, // Ampere 세대 (SM 8.0) GA100 클래스
        { 0x86, 128 }, // Ampere 세대 (SM 8.6) GA10x 클래스
        { -1, -1 }
    };

    int index = 0;

    while (nGpuArchCoresPerSM[index].SM != -1) {
        if (nGpuArchCoresPerSM[index].SM == ((major << 4) + minor)) {
            return nGpuArchCoresPerSM[index].Cores;
        }
        index++;
    }

    // 값을 찾지 못한 경우, 이전 값을 사용하여 제대로 실행되도록 기본값을 사용
    printf("SM %d.%d에 대한 MapSMtoCores가 정의되지 않았습니다. 기본값으로 %d Cores/SM을 사용합니다\n", major, minor, nGpuArchCoresPerSM[13].Cores);
    return nGpuArchCoresPerSM[13].Cores;
}

int main(void)
{
    int ngpus;
    hipGetDeviceCount(&ngpus);

    for (int i = 0; i < ngpus; i++)
    {
        hipDeviceProp_t devProp;
        hipGetDeviceProperties(&devProp, i);

        printf("Device %d: %s\n", i, devProp.name);
        printf("\tCompute Capability: %d.%d\n", devProp.major, devProp.minor);
        printf("\tCUDA 코어 수: %d\n", ConvertSMVer2Cores(devProp.major, devProp.minor) * devProp.multiProcessorCount);
        printf("\t글로벌 메모리 크기: %.2f MB\n", (float)devProp.totalGlobalMem / _1MB);
    }

    return 0;
}