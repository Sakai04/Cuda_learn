
#include <hip/hip_runtime.h>
/*#include "cuda_runtime.h"
#include "device_launch_parameters.h"


#include <stdio.h>
#include <stdlib.h>
#include <string.h>

// The size of vector
#define NUM_DATA 1024

// Simple vector sum kernel
__global__ void vecAdd(int* _a, int* _b, int* _c)
{
    int tID = threadIdx.x;
    _c[tID] = _a[tID] + _b[tID];
}

int main(void)
{
    
    int* a, * b, * c, * hc; // vectors on the host
    int* da, * db, * dc; // vectors on the device

    int memSize = sizeof(int) * NUM_DATA;
    printf("%d elements, memSize = %d bytes\n", NUM_DATA, memSize);

    // memory allocation on the host-side
    a = new int[NUM_DATA]; memset(a, 0, memSize);
    b = new int[NUM_DATA]; memset(b, 0, memSize);
    c = new int[NUM_DATA]; memset(c, 0, memSize);
    hc = new int[NUM_DATA]; memset(hc, 0, memSize);

    // data generation
    for (int i = 0; i < NUM_DATA; i++)
    {
        a[i] = rand() % 10;
        b[i] = rand() % 10;
    }

    // vector sum on host
    for (int i = 0; i < NUM_DATA; i++)
        hc[i] = a[i] + b[i];

    // memory allocation on the device-side
    cudaMalloc(&da, memSize); cudaMemset(da, 0, memSize);
    cudaMalloc(&db, memSize); cudaMemset(db, 0, memSize);
    cudaMalloc(&dc, memSize); cudaMemset(dc, 0, memSize);

    // data copy : host to device
    cudaMemcpy(da, a, memSize, cudaMemcpyHostToDevice);
    cudaMemcpy(db, b, memSize, cudaMemcpyHostToDevice);

    // kernel call
    vecAdd<<<1, NUM_DATA>>>(da, db, dc);

    // copy results : device to host
    cudaMemcpy(c, dc, memSize, cudaMemcpyDeviceToHost);

    // release device memory
    cudaFree(da); cudaFree(db); cudaFree(dc);

    // check results
    bool result = true;
    for (int i = 0; i < NUM_DATA; i++)
    {
        if (hc[i] != c[i])
        {
            printf("[%d] the result is not matched! (%d, %d)\n", i, hc[i], c[i]);
            result = false;
        }
    }

    if (result)
        printf("gpu works well\n");

    // release host memory
    delete[] a; delete[] b; delete[] c; delete[] hc;

    return 0;
}*/