
#include <hip/hip_runtime.h>
/*#include "cuda_runtime.h"
#include "device_launch_parameters.h"
#include <stdio.h>

#define ALLOC_SIZE (1024 * 1024 * sizeof(int))

void check_device_memory(void)
{
    size_t free, total;
    cudaMemGetInfo(&free, &total);
    printf("Device memory (free/total = %zu/%zu bytes)\n", free, total);
}

void check_cuda_error(cudaError_t errorCode, const char* action)
{
    if (errorCode != cudaSuccess)
    {
        fprintf(stderr, "Error during %s: %s\n", action, cudaGetErrorString(errorCode));
    }
    else
    {
        printf("%s - %s\n", action, cudaGetErrorName(errorCode));
    }
}

int main(void)
{
    int* dDataPtr;
    cudaError_t errorCode;

    check_device_memory();

    errorCode = cudaMalloc(&dDataPtr, ALLOC_SIZE);
    check_cuda_error(errorCode, "cudaMalloc");
    check_device_memory();

    errorCode = cudaMemset(dDataPtr, 0, ALLOC_SIZE);
    check_cuda_error(errorCode, "cudaMemset");
    check_device_memory();

    errorCode = cudaFree(dDataPtr);
    check_cuda_error(errorCode, "cudaFree");
    check_device_memory();

    return 0;
}*/
